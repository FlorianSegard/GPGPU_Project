#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "backgroundestimationfirsttry/labConverter.hpp" //maybe do it better like do a library in the makefile or somehtign
#include "filter_impl.h"

// Cuda error checking macro
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Separate kernel launch error checking function
inline void checkKernelLaunch() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel synchronization error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ============== CUDA FUNCTIONS ==============








size_t background_ref_pitch;
lab* background_ref = nullptr;
size_t candidate_bg_pitch;
lab* candidate_background = nullptr;

// TODO: what to do when background_ref / candidate_background null?
// TODO: is it possible to reuse buffers instead of always creating new ones?

void filter_impl(uint8_t* pixels_buffer, int width, int height, int plane_stride, int pixel_stride)
{


    Parameters params;    
    params.device = GPU;

    // GPU properties for kernel calls
    hipError_t error;
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Alloc memory and copy input RGB buffer
    // -> hipMemcpy2D 'kind' param - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g18fa99055ee694244a270e4d5101e95b
    
    
    // size_t rgb_pitch;
    // rgb8* rgb_buffer; // type: rgb8 array pointer
    // error = hipMallocPitch(&rgb_buffer, &rgb_pitch,
    //                         width * sizeof(rgb8), height);

    // CHECK_CUDA_ERROR(error);

    Image<rgb8> rgb_image(width, height, true);


    error = hipMemcpy2D(rgb_image.buffer, rgb_image.stride, pixels_buffer, plane_stride,
                         width * sizeof(rgb8), height, hipMemcpyDefault);

    CHECK_CUDA_ERROR(error);

    // Allocate LAB converted image buffer

    // size_t lab_pitch;
    // lab* lab_buffer; // type: lab array pointer

    // error = hipMallocPitch(&lab_buffer, &lab_pitch,
    //                         width * sizeof(lab), height);
    // CHECK_CUDA_ERROR(error);

    Image<lab> lab_image(width, height, true);


    // Convert RGB to LAB

    labConv_init(&params);

    labConv_process_frame(rgb_image, lab_image);

    checkKernelLaunch();

    // Residual image
    size_t residual_pitch;
    lab* residual_buffer; // type: lab array pointer
    error = hipMallocPitch(&residual_buffer, &residual_pitch,
                            width * sizeof(lab), height);
    CHECK_CUDA_ERROR(error);

    // TODO: GPU residual image to code with the following args
    // - background_ref, background_ref_pitch     : the background reference
    // - lab_buffer, lab_pitch                    : the current image
    // - residual_buffer, residual_pitch          : the buffer to fill
    // - heigt and width





    // residual_image<<<blocksPerGrid, threadsPerBlock>>>();


    // checkKernelLaunch();





    // // Update background model
    // check_background_GPU<<<blocksPerGrid, threadsPerBlock>>>(
    //     lab_buffer, lab_pitch,
    //     background_ref, background_ref_pitch,
    //     candidate_background, candidate_bg_pitch,
    //     (int*)current_time_pixels, time_pixels_pitch,
    //     width, height
    // );
    // checkKernelLaunch();

    // // Perform eroding operation
    // size_t eroded_pitch;
    // lab* eroded_buffer; // type: lab array pointer
    // error = hipMallocPitch(&eroded_buffer, &eroded_pitch,
    //                         width * sizeof(lab), height);
    // CHECK_CUDA_ERROR(error);

    // erode<<<blocksPerGrid, threadsPerBlock>>>(
    //     residual_buffer, eroded_buffer,
    //     width, height, residual_pitch
    // );
    // checkKernelLaunch();

    // // Perform dilatation operation
    // size_t dilated_pitch;
    // lab* dilated_buffer; // type: lab array pointer
    // error = hipMallocPitch(&dilated_buffer, &dilated_pitch,
    //                         width * sizeof(lab), height);
    // CHECK_CUDA_ERROR(error);

    // dilate<<<blocksPerGrid, threadsPerBlock>>>(
    //     eroded_buffer, dilated_buffer,
    //     width, height, eroded_pitch
    // );
    // checkKernelLaunch();

    // // Perform hysteresis operation
    // size_t hysteresis_pitch;
    // bool* hysteresis_buffer; // type: bool array pointer
    // error = hipMallocPitch(&hysteresis_buffer, &hysteresis_pitch,
    //                         width * sizeof(bool), height);
    // CHECK_CUDA_ERROR(error);

    // hysteresis_reconstruction<<<blocksPerGrid, threadsPerBlock>>>(
    //     dilated_buffer, hysteresis_buffer,
    //     width, height, dilated_pitch
    // );
    // checkKernelLaunch();

    // // TODO: Apply the new created hysteresis mask to rgb_buffer
    // // - hysteresis_buffer, hysteresis_pitch      : the mask buffer
    // // - rgb_buffer, rgb_pitch                    : the buffer to change
    // // - heigt and widt h
    // apply_mask<<<blocksPerGrid, threadsPerBlock>>>();
    // checkKernelLaunch();

    // // Copy result back to pixels_buffer
    // error = hipMemcpy2D(pixels_buffer, plane_stride, rgb_buffer, rgb_pitch,
    //                      width * sizeof(rgb8), height, hipMemcpyDeviceToHost);
    // CHECK_CUDA_ERROR(error);

    // // Clean up temporary buffers
    // hipFree(rgb_buffer);
    // hipFree(lab_buffer);
    // hipFree(residual_buffer);
    // hipFree(eroded_buffer);
    // hipFree(dilated_buffer);
}