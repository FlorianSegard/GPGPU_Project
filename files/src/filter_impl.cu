#include <iostream>

struct rgb {
    uint8_t r;
    uint8_t g;
    uint8_t b;
};

struct lab {
    float L;
    float a;
    float b;
};

// Cuda error checking macro
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

size_t background_ref_pitch;
std::byte* background_ref = nullptr;
size_t candidate_bg_pitch;
std::byte* candidate_background = nullptr;

// TODO: what to do when background_ref / candidate_background null?
// TODO: is it possible to reuse buffers instead of always creating new ones?


extern "C"
{
    void filter_impl(uint8_t* pixels_buffer, int width, int height, int plane_stride, int pixel_stride, GstClockTime timestamp)
    {
        // GPU properties for kernel calls
        hipError_t error;
        dim3 threadsPerBlock(32, 32);
        dim3 blocksPerGrid((width + blockSize.x - 1) / blockSize.x,
                           (height + blockSize.y - 1) / blockSize.y);

        // Alloc memory and copy input RGB buffer
        // -> hipMemcpy2D 'kind' param - https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g18fa99055ee694244a270e4d5101e95b
        size_t pitch;
        std::byte* rgb_buffer; // type: rgb array pointer
        error = hipMallocPitch(&rgb_buffer, &pitch,
                                width * sizeof(rgb), height);
        CHECK_CUDA_ERROR(error);
        error = hipMemcpy2D(rgb_buffer, pitch, pixels_buffer, plane_stride,
                             width * sizeof(rgb), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(error);

        // Allocate LAB converted image buffer
        size_t lab_pitch;
        std::byte* lab_buffer; // type: lab array pointer
        error = hipMallocPitch(&lab_buffer, &lab_pitch,
                                width * sizeof(lab), height));
        CHECK_CUDA_ERROR(error)

        // Convert RGB to LAB
        rgbtolab_converter_GPU<<<blocksPerGrid, threadsPerBlock>>>(
            (rgb8*)rgb_buffer, rgb_pitch,
            (lab*)lab_buffer, lab_pitch,
            width, height
        )
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Residual image
        size_t residual_pitch;
        std::byte* residual_buffer; // type: lab array pointer
        error = hipMallocPitch(&residual_buffer, &residual_pitch,
                                width * sizeof(Lab), height);
        CHECK_CUDA_ERROR(error);

        // TODO: GPU residual image to code with the following args
        // - background_ref, background_ref_pitch     : the background reference
        // - lab_buffer, lab_pitch                    : the current image
        // - residual_buffer, residual_pitch          : the buffer to fill
        // - heigt and width
        residual_image<<<blocksPerGrid, threadsPerBlock>>>();
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Update background model
        check_background_GPU<<<blocksPerGrid, threadsPerBlock>>>(
            (lab*)lab_buffer, lab_pitch,
            (lab*)background_ref, background_ref_pitch,
            (lab*)candidate_background, candidate_bg_pitch,
            (int*)current_time_pixels, time_pixels_pitch,
            width, height
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Perform eroding operation
        size_t eroded_pitch;
        std::byte* eroded_buffer; // type: lab array pointer
        error = hipMallocPitch(&eroded_buffer, &eroded_pitch,
                                width * sizeof(lab), height);
        CHECK_CUDA_ERROR(error);

        erode<<<blocksPerGrid, threadsPerBlock>>>(
            (lab*)residual_buffer, (lab*)eroded_buffer,
            width, height, residual_pitch
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Perform dilatation operation
        size_t dilated_pitch;
        std::byte* dilated_buffer; // type: lab array pointer
        error = hipMallocPitch(&dilated_buffer, &dilated_pitch,
                                width * sizeof(lab), height);
        CHECK_CUDA_ERROR(error);

        dilate<<<blocksPerGrid, threadsPerBlock>>>(
            (lab*)eroded_buffer, (lab*)dilated_buffer,
            width, height, eroded_pitch
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Perform hysteresis operation
        size_t hysteresis_pitch;
        std::byte* hysteresis_buffer; // type: bool array pointer
        error = hipMallocPitch(&hysteresis_buffer, &hysteresis_pitch,
                                width * sizeof(bool), height);
        CHECK_CUDA_ERROR(error);

        hysteresis_reconstruction<<<blocksPerGrid, threadsPerBlock>>>(
            (lab*)dilated_buffer, (lab*)hysteresis_buffer,
            width, height, dilated_pitch
        );
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // TODO: Apply the new created hysteresis mask to rgb_buffer
        // - hysteresis_buffer, hysteresis_pitch      : the mask buffer
        // - rgb_buffer, rgb_pitch                    : the buffer to change
        // - heigt and widt h
        apply_mask<<<blocksPerGrid, threadsPerBlock>>>();
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Copy result back to pixels_buffer
        error = hipMemcpy2D(pixels_buffer, plane_stride, rgb_buffer, rgb_pitch,
                             width * sizeof(rgb8), height, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(error)

        // Clean up temporary buffers
        hipFree(rgb_buffer);
        hipFree(lab_buffer);
        hipFree(residual_buffer);
        hipFree(eroded_buffer);
        hipFree(dilated_buffer);
        hipFree(marker_buffer);
        hipFree(output_buffer);
    }
}