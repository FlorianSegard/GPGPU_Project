#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "logic/labConverter.hpp"
#include "logic/backgroundestimation.hpp"
#include "logic/filter_erode_and_dilate.hpp"
#include "logic/hysteresis.hpp"
#include "logic/red_mask.hpp"
#include "filter_impl.h"

// Cuda error checking macro
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Separate kernel launch error checking function
inline void checkKernelLaunch() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel synchronization error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ============== CUDA FUNCTIONS ==============

__global__ void debug_bool_kernel(ImageView<bool> bf, ImageView<rgb8> rgb_buffer, int width, int height, std::ptrdiff_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    bool bl = (bool*)((std::byte*)bf.buffer + y * bf.stride)[x];
    rgb8* rgb_value = (rgb8*)((std::byte*)rgb_buffer.buffer + y * rgb_buffer.stride);

    rgb_value[x].r = bl ? 255 : 0;//rgb_value[x].r / 2 + (bf ? 127 : 0);
    rgb_value[x].g = bl ? 255 : 0;//rgb_value[x].g / 2;
    rgb_value[x].b = bl ? 255 : 0;//rgb_value[x].b / 2;
}


__global__ void debug_float_kernel(ImageView<float> bf, ImageView<rgb8> rgb_buffer, int width, int height, std::ptrdiff_t stride) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float* bl = (float*)((std::byte*)bf.buffer + y * bf.stride);
    rgb8* rgb_value = (rgb8*)((std::byte*)rgb_buffer.buffer + y * rgb_buffer.stride);

    rgb_value[x].r = round(fmaxf((int)(bl[x]), 255.0));
    rgb_value[x].g = round(fmaxf((int)(bl[x]), 255.0));
    rgb_value[x].b = round(fmaxf((int)(bl[x]), 255.0));
}


Image<lab> current_background;
Image<lab> candidate_background;
Image<int> current_time_pixels;
bool isInitialized = false;

void initializeGlobals(int width, int height) {
    if (!isInitialized) {
        current_background = Image<lab>(width, height, true);
        candidate_background = Image<lab>(width, height, true);
        current_time_pixels = Image<int>(width, height, true);
        isInitialized = true;
    }
}

// TODO: what to do when background_ref / candidate_background null?
// TODO: is it possible to reuse buffers instead of always creating new ones?
// Check error after each initialization
extern "C" {
void filter_impl_cu(uint8_t* pixels_buffer, int width, int height, int plane_stride)
{
    // Init device and global variables
    Parameters params;
    params.device = GPU;
    initializeGlobals(width, height);



    // GPU properties for kernel calls
    hipError_t error;
    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Clone pixels_buffer inside new allocated rgb_buffer
    Image<rgb8> rgb_image(width, height, true);
    error = hipMemcpy2D(rgb_image.buffer, rgb_image.stride, pixels_buffer, plane_stride,
                         width * sizeof(rgb8), height, hipMemcpyDefault);
    CHECK_CUDA_ERROR(error);



    // Allocate lab converted image buffer
    labConv_init(&params);
    Image<lab> lab_image(width, height, true);

    // Convert RGB to LAB -> result stored inside lab_buffer
    labConv_process_frame(rgb_image, lab_image);
    hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "labConv call succeeded" << std::endl;



    // Update background and get residual image
    background_init(&params);
    Image<float> residual_image(width, height, true);

    background_process_frame(lab_image, current_background, candidate_background, current_time_pixels, residual_image);
	hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "background call succeeded" << std::endl;



    // Alloc and perform eroding operation
    filter_init(&params);
    Image<float> erode_image(width, height, true);

    erode_process_frame(
            residual_image, erode_image,
         width, height, plane_stride
    );
    hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "erode call succeeded" << std::endl;



    // Alloc and perform eroding operation
    Image<float> dilate_image(width, height, true);

    dilate_process_frame(
            erode_image, dilate_image,
            width, height, plane_stride
    );
    hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "dilate call succeeded" << std::endl;

    debug_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(dilate_image, rgb_image, width, height, plane_stride);
    /*

    // Alloc and perform hysteresis operation
    hysteresis_init(&params);
    Image<bool> hysteresis_image(width, height, true);

    //TODO: retrieve threshold values
    hysteresis_process_frame(
            dilate_image, hysteresis_image,
            width, height, 3, 30
    );
    hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "hysteresis call succeeded" << std::endl;


    // Alloc and red mask operation
    mask_process_frame(hysteresis_image, rgb_image, width, height, plane_stride);
    hipDeviceSynchronize();
    checkKernelLaunch();
    std::cout << "red mask call succeeded" << std::endl;

    */



    // // Copy result back to pixels_buffer
    error = hipMemcpy2D(pixels_buffer, plane_stride, rgb_image.buffer, rgb_image.stride,
                         width * sizeof(rgb8), height, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR(error);
    std::cout << "copy back to pixels_buffer" << std::endl;

    // // Clean up temporary buffers
    // hipFree(rgb_buffer);
    // hipFree(lab_buffer);
    // hipFree(residual_buffer);
    // hipFree(eroded_buffer);
    // hipFree(dilated_buffer);
}
}