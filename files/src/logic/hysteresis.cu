#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include "hysteresis.hpp"

// Helper function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define HYSTERESIS_TILE_WIDTH 34 // block size de 32 x 32 et on rajoute 2 pixels de padding
#define LOWER_THRESHOLD 4.0
#define UPPER_THRESHOLD 30.0

// -----------------------------------------------------------

__global__ void hysteresis_thresholding(ImageView<float> input, ImageView<bool> output, int width, int height, float threshold)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float *input_lineptr = (float *)((std::byte*)input.buffer + y * input.stride);
    float in_val = input_lineptr[x];

    // Applique le seuil et on stocke le résultat dans la sortie
    bool *output_lineptr = (bool *)((std::byte*)output.buffer + y * output.stride);
    output_lineptr[x] = in_val > threshold;
}


__global__ void hysteresis_kernel(ImageView<bool> upper, ImageView<bool> lower, int width, int height, bool *has_changed_global)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    bool has_changed = true;

    while (has_changed)
    {
        has_changed = false;
        __syncthreads();

        bool *upper_lineptr = (bool *)((std::byte*)upper.buffer + y * upper.stride);
        bool *lower_lineptr = (bool *)((std::byte*)lower.buffer + y * lower.stride);

        // Si le pixel est déjà marqué dans l'image supérieure, on passe au suivant
        if (upper_lineptr[x])
            break;

        // Si le pixel n'est pas marqué dans l'image inférieure, on passe au suivant
        if (!lower_lineptr[x])
            break;

        // on vérifie les pixels voisins pour propager le marquage
        if ((x > 0 && upper_lineptr[x - 1]) ||
            (x < width - 1 && upper_lineptr[x + 1]) ||
            (y > 0 && ((bool *)((std::byte*)upper.buffer + (y - 1) * upper_pitch))[x]) ||
            (y < height - 1 && ((bool *)((std::byte*)upper.buffer + (y + 1) * upper_pitch))[x]))
        {
            upper_lineptr[x] = true;
            has_changed = true;
            *has_changed_global = true;
            break;
        }

        __syncthreads();
    }
}

void hysteresis_cu(ImageView<float> opened_input, ImageView<bool> hysteresis, int width, int height, float lower_threshold, float upper_threshold)
{
    dim3 blockSize(32, 32);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

    Image<bool> lower_threshold_input(width, height, true);

    // seuil inf et sup
    hysteresis_thresholding<<<gridSize, blockSize>>>(opened_input, lower_threshold_input, width, height, lower_threshold);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    hysteresis_thresholding<<<gridSize, blockSize>>>(opened_input, hysteresis, width, height, upper_threshold);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    bool h_has_changed = true;

    // flag de changement
    bool *d_has_changed;
    CHECK_CUDA_ERROR(hipMalloc(&d_has_changed, sizeof(bool)));

    // on propage sur l'image.
    while (h_has_changed)
    {
        CHECK_CUDA_ERROR(hipMemset(d_has_changed, false, sizeof(bool)));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        hysteresis_kernel<<<gridSize, blockSize>>>(hysteresis, lower_threshold_input, width, height, d_has_changed);

        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        CHECK_CUDA_ERROR(hipMemcpy(&h_has_changed, d_has_changed, sizeof(bool), hipMemcpyDeviceToHost));
    }

    hipFree(d_has_changed);
}
