#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include "hysteresis.hpp"

// Helper function to check CUDA errors
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define BLOCK_SIZE 30
#define HYSTERESIS_TILE_WIDTH (BLOCK_SIZE + 2)
#define LOWER_THRESHOLD 4.0
#define UPPER_THRESHOLD 30.0


__global__ void hysteresis_thresholding(ImageView<float> input, ImageView<bool> output, int width, int height, float threshold)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int tile_x = blockIdx.x * blockDim.x;
    int tile_y = blockIdx.y * blockDim.y;

    int x = tile_x + tx;
    int y = tile_y + ty;

    if (x >= width || y >= height)
        return;

    // On charge la tuile
    float* input_lineptr = (float *)((std::byte*)input.buffer + y * input.stride);

    // On applique le seuil
    bool out_val = input_lineptr[x] > threshold;

    // On stocke le résultat dans la sortie
    bool *output_lineptr = (bool *)((std::byte*)output.buffer + y * output.stride);
    output_lineptr[x] = out_val;
}

__global__ void hysteresis_kernel(ImageView<bool> upper, ImageView<bool> lower, int width, int height, bool *has_changed_global)
{
    __shared__ bool tile_upper[HYSTERESIS_TILE_WIDTH][HYSTERESIS_TILE_WIDTH];
    __shared__ bool tile_lower[HYSTERESIS_TILE_WIDTH][HYSTERESIS_TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate global coordinates adjusted for halo
    int x = blockIdx.x * BLOCK_SIZE + tx - 1;
    int y = blockIdx.y * BLOCK_SIZE + ty - 1;

    // Load data into shared memory with boundary checks
    bool upper_value = false;
    bool lower_value = true;
    
    bool* upper_lineptr = (bool *)((std::byte*)upper.buffer + y * upper.stride);

    if (x >= 0 && x < width && y >= 0 && y < height)
    {
        bool* lower_lineptr = (bool *)((std::byte*)lower.buffer + y * lower.stride);
        upper_value = upper_lineptr[x];
        lower_value = lower_lineptr[x];
    }

    tile_upper[ty][tx] = upper_value;
    tile_lower[ty][tx] = lower_value;

    if (x >= width - 1 || y >= height - 1 || x == 0 || y == 0)
        return;

    if (tile_upper[ty][tx])
        return;

    if (!tile_lower[ty][tx])
        return;

    __syncthreads();

    // Only process inner pixels
    if (tx > 0 && tx < HYSTERESIS_TILE_WIDTH - 1 && ty > 0 && ty < HYSTERESIS_TILE_WIDTH - 1)
    {

        if (tile_upper[ty][tx - 1])
        {
            upper_lineptr[x] = true;
            *has_changed_global = true;
        }

        if (tile_upper[ty][tx + 1])
        {
            upper_lineptr[x] = true;
            *has_changed_global = true;
        }
        if (tile_upper[ty - 1][tx])
        {
            upper_lineptr[x] = true;
            *has_changed_global = true;
        }
        if (tile_upper[ty + 1][tx])
        {
            upper_lineptr[x] = true;
            *has_changed_global = true;
        }
        return;
    }
    if (upper_lineptr[x - 1])
    {
        upper_lineptr[x] = true;
        *has_changed_global = true;
    }
    if (upper_lineptr[x - 1])
    {
        upper_lineptr[x] = true;
        *has_changed_global = true;
    }
    if ((bool *)((std::byte*)upper.buffer + (y - 1) * upper.stride)[x])
    {
        upper_lineptr[x] = true;
        *has_changed_global = true;
    }
    if ((bool *)((std::byte*)upper.buffer + (y + 1) * upper.stride)[x])
    {
        upper_lineptr[x] = true;
        *has_changed_global = true;
    }

}



void hysteresis_cu(ImageView<float> opened_input, ImageView<bool> hysteresis, int width, int height, float lower_threshold, float upper_threshold)
{
    dim3 blockSize(32, 32);
    dim3 gridSize((width + (blockSize.x - 1)) / blockSize.x, (height + (blockSize.y - 1)) / blockSize.y);

    Image<bool> lower_threshold_input(width, height, true);

    // seuil inf et sup
    hysteresis_thresholding<<<gridSize, blockSize>>>(opened_input, lower_threshold_input, width, height, lower_threshold);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    hysteresis_thresholding<<<gridSize, blockSize>>>(opened_input, hysteresis, width, height, upper_threshold);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    bool h_has_changed = 1;

    // flag de changement
    bool *d_has_changed;
    CHECK_CUDA_ERROR(hipMalloc(&d_has_changed, sizeof(bool)));


    // on propage sur l'image.
    while (h_has_changed)
    {
        CHECK_CUDA_ERROR(hipMemset(d_has_changed, false, sizeof(bool)));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        hysteresis_kernel<<<gridSize, blockSize>>>(hysteresis, lower_threshold_input, width, height, d_has_changed);

        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        CHECK_CUDA_ERROR(hipMemcpy(&h_has_changed, d_has_changed, sizeof(bool), hipMemcpyDeviceToHost));
    }

    //printf("%d\n", i);
    hipFree(d_has_changed);
}

