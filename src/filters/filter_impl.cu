#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include "../logic/lab_converter/lab_converter.hpp"
#include "../logic/background/background_estimation.hpp"
#include "erode_and_dilate/filter_erode_and_dilate.hpp"
#include "../logic/hysteresis/hysteresis.hpp"
#include "../logic/red_mask/red_mask.hpp"
#include "filter_impl.hpp"

// Cuda error checking macro
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// Separate kernel launch error checking function
inline void checkKernelLaunch(bool is_gpu) {
    if (!is_gpu)
        return;

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel synchronization error: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ============== CUDA FUNCTIONS FOR DEBUG ==============

// GPU properties for cuda debug purpose kernel calls
//hipError_t error;
//dim3 threadsPerBlock(32, 32);
//dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
//                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

__global__ void debug_bool_kernel(ImageView<bool> bf, ImageView<rgb8> rgb_buffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    bool bl = (bool*)((std::byte*)bf.buffer + y * bf.stride)[x];
    rgb8* rgb_value = (rgb8*)((std::byte*)rgb_buffer.buffer + y * rgb_buffer.stride);

    rgb_value[x].r = bl ? 255 : 0;//rgb_value[x].r / 2 + (bf ? 127 : 0);
    rgb_value[x].g = bl ? 255 : 0;//rgb_value[x].g / 2;
    rgb_value[x].b = bl ? 255 : 0;//rgb_value[x].b / 2;
}


__global__ void debug_float_kernel(ImageView<float> bf, ImageView<rgb8> rgb_buffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    float* bl = (float*)((std::byte*)bf.buffer + y * bf.stride);
    rgb8* rgb_value = (rgb8*)((std::byte*)rgb_buffer.buffer + y * rgb_buffer.stride);

    rgb_value[x].r = (uint8_t) round(fminf((bl[x]), 255.0));
    rgb_value[x].g = (uint8_t) round(fminf((bl[x]), 255.0));
    rgb_value[x].b = (uint8_t) round(fminf((bl[x]), 255.0));
}

// ============== MAIN IMAGE PROCESSING ==============

Image<lab> current_background;
Image<lab> candidate_background;
Image<int> current_time_pixels;
bool isInitialized = false;

void initializeGlobals(int width, int height, ImageView<lab> lab_image, bool is_gpu) {
    current_background = Image<lab>(width, height, is_gpu);
    candidate_background = Image<lab>(width, height, is_gpu);
    current_time_pixels = Image<int>(width, height, is_gpu);
    isInitialized = true;

    if (is_gpu) {
        hipError_t error;
        error = hipMemcpy2D(current_background.buffer, current_background.stride, lab_image.buffer, lab_image.stride,
                             width * sizeof(lab), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(error);
        error = hipMemcpy2D(candidate_background.buffer, candidate_background.stride, lab_image.buffer,
                             lab_image.stride,
                             width * sizeof(lab), height, hipMemcpyDefault);
        CHECK_CUDA_ERROR(error);
        std::cout << "Running on GPU" << std::endl;
    }
    else {
        for (int y = 0; y < lab_image.height; ++y)
            memcpy((char*)current_background.buffer + y * current_background.stride,
                   (char*)lab_image.buffer + y * lab_image.stride,
                   lab_image.width * sizeof(lab));

        for (int y = 0; y < lab_image.height; ++y)
            memcpy((char*)candidate_background.buffer + y * candidate_background.stride,
                   (char*)lab_image.buffer + y * lab_image.stride,
                   lab_image.width * sizeof(lab));

        std::cout << "Running on CPU" << std::endl;
    }
}


extern "C" {
    void filter_impl(uint8_t* pixels_buffer, int width, int height, int plane_stride, e_device_t device,
                        const char* bg_uri, int opening_size, int th_low, int th_high, int bg_sampling_rate, int bg_number_frame)
    {
        // Init device and device variables
        Parameters params;
        params.device = device;
        bool is_gpu = device == GPU;

        hipError_t error;
        lab_conv_init(&params);
        background_init(&params);
        filter_init(&params);
        hysteresis_init(&params);
        mask_init(&params);


        // Clone pixels_buffer inside new allocated rgb_buffer
        Image<rgb8> rgb_image(width, height, is_gpu);
        if (is_gpu) {
            error = hipMemcpy2D(rgb_image.buffer, rgb_image.stride, pixels_buffer, plane_stride,
                                 width * sizeof(rgb8), height, hipMemcpyDefault);
            CHECK_CUDA_ERROR(error);
        }
        else {
            for (int y = 0; y < rgb_image.height; ++y)
                memcpy((char*)rgb_image.buffer + y * rgb_image.stride,
                        (char*)pixels_buffer + y * plane_stride,
                        rgb_image.width * sizeof(rgb8));
        }


        // Allocate lab converted image buffer
        Image<lab> lab_image(width, height, is_gpu);

        // Convert RGB to LAB -> result stored inside lab_buffer
        lab_conv_process_frame(rgb_image, lab_image);
        checkKernelLaunch(is_gpu);

        if (!isInitialized)
            initializeGlobals(width, height, lab_image, is_gpu);

        // Update background and get residual image
        Image<float> residual_image(width, height, is_gpu);

        background_process_frame(lab_image, current_background, candidate_background,
                                 current_time_pixels, residual_image, bg_number_frame);
        checkKernelLaunch(is_gpu);
        //debug_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(residual_image, rgb_image, width, height);


        // Alloc and perform eroding operation
        Image<float> erode_image(width, height, is_gpu);
        erode_process_frame(
                residual_image, erode_image,
                width, height, opening_size / 2
        );
        checkKernelLaunch(is_gpu);
        //debug_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(erode_image, rgb_image, width, height);


        // Keep old residual_image alloc and perform dilatation operation
        dilate_process_frame(
                erode_image, residual_image,
                width, height, opening_size / 2
        );
        checkKernelLaunch(is_gpu);
        //debug_float_kernel<<<blocksPerGrid, threadsPerBlock>>>(dilate_image, rgb_image, width, height);


        // Alloc and perform hysteresis operation
        Image<bool> hysteresis_image(width, height, is_gpu);
        hysteresis_process_frame(
                residual_image, hysteresis_image,
                width, height, th_low, th_high
        );
        checkKernelLaunch(is_gpu);
        //debug_bool_kernel<<<blocksPerGrid, threadsPerBlock>>>(hysteresis_image, rgb_image, width, height);


        // Alloc and red mask operation
        mask_process_frame(hysteresis_image, rgb_image, width, height);
        checkKernelLaunch(is_gpu);
        std::cout << "Running on 5" << std::endl;


        // Copy result back to pixels_buffer
        if (is_gpu) {
            error = hipMemcpy2D(pixels_buffer, plane_stride, rgb_image.buffer, rgb_image.stride,
                                 width * sizeof(rgb8), height, hipMemcpyDeviceToHost);
            CHECK_CUDA_ERROR(error);
        }
        else {
            for (int y = 0; y < rgb_image.height; ++y)
                memcpy(pixels_buffer + y * plane_stride,
                       (char*)rgb_image.buffer + y * rgb_image.stride,
                       rgb_image.width * sizeof(rgb8));
        }
    }
}